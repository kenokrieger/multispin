#include "hip/hip_runtime.h"
#pragma clang diagnostic push
#pragma ide diagnostic ignored "cppcoreguidelines-narrowing-conversions"
#include <cstdio>
#include <string>
#include <iostream>
#include <fstream>
#include <map>

#include "cudamacro.h"
#include "traders.cuh"

using namespace std;

#define DIV_UP(a,b)  (((a) + ((b) - 1)) / (b))
#define MIN(a,b)	(((a) < (b)) ? (a) : (b))

#define THREADS (128)
#define BIT_X_SPIN (4)

#define THREADS_X (16)
#define THREADS_Y (16)

#define FILE_ENTRY_LIMIT (1000000)


map<string, string> read_config_file(const string& config_filename, const string& delimiter = "=") {
    std::ifstream config_file;
    config_file.open(config_filename);
    map<string, string> config;

    if (!config_file.is_open()) {
        std::cout << "Could not open config file '" << config_filename << "'" << std::endl;
        exit(EXIT_FAILURE);
    }

    int row = 0;
    std::string line;
    std::string key;

    while (getline(config_file, line)) {
        if (line[0] == '#' || line.empty()) continue;
        int delimiter_position = line.find(delimiter);

        for (int idx = 0; idx < delimiter_position; idx++) {
            if (line[idx] != ' ') key += line[idx];
        }

        std::string value = line.substr(delimiter_position + 1, line.length() - 1);
        config[key] = value;
        row++;
        key = "";
    }
    config_file.close();
    return config;
}


void validate_grid(const long long lattice_width, const long long lattice_height,
                   const int spin_x_word) {
    if (!lattice_width || (lattice_width % 2) || ((lattice_width / 2) % (2 * spin_x_word * THREADS_X))) {
        fprintf(stderr, "\nPlease specify an lattice_width multiple of %d\n\n", 2 * spin_x_word * 2 * THREADS_X);
        exit(EXIT_FAILURE);
    }
    if (!lattice_height || (lattice_height % (THREADS_Y))) {
        fprintf(stderr, "\nPlease specify a lattice_height multiple of %d\n\n", THREADS_Y);
        exit(EXIT_FAILURE);
    }
}


hipDeviceProp_t identify_gpu() {
    hipDeviceProp_t props{};
    CHECK_CUDA(hipGetDeviceProperties(&props, 0))
    /*
    printf("\nUsing GPU: %s, %d SMs, %d th/SM max, CC %d.%d, ECC %s\n",
    props.name, props.multiProcessorCount,
    props.maxThreadsPerMultiProcessor,
    props.major, props.minor,
    props.ECCEnabled ? "on" : "off");
    */
    return props;
}


int main(int argc, char **argv) {
    unsigned long long *d_spins = nullptr;
    const int SPIN_X_WORD = (8 * sizeof(*d_spins)) / BIT_X_SPIN;
    unsigned long long *d_black_tiles;
    unsigned long long *d_white_tiles;

    unsigned long long spins_up;
    unsigned long long spins_down;
    unsigned long long *d_sum;

    hipEvent_t start, stop;
    float elapsed_time;

    std::ofstream mag_file;
    Parameters params;

    string config_filename = (argc == 1) ? "multising.conf" : argv[1];
    map<string, string> config = read_config_file(config_filename);

    params.lattice_height = std::stoll(config["lattice_height"]);
    params.lattice_width = std::stoll(config["lattice_width"]);
    params.seed = std::stoull(config["seed"]);
    const unsigned int total_updates = std::stoul(config["total_updates"]);
    float alpha = std::stof(config["alpha"]);
    float j = std::stof(config["j"]);
    float beta = std::stof(config["beta"]);
    float percentage_up = std::stof(config["init_up"]);

    params.reduced_alpha = -2.0f * beta * alpha;
    params.reduced_j = -2.0f * beta * j;

    validate_grid(params.lattice_width, params.lattice_height, SPIN_X_WORD);
    hipDeviceProp_t props = identify_gpu();

    params.words_per_row = (params.lattice_width / 2) / SPIN_X_WORD;
    params.total_words = 2ull * static_cast<size_t>(params.lattice_height) * params.words_per_row;

    // words_per_row / 2 because each entry in the array has two components
    dim3 blocks(DIV_UP(params.words_per_row / 2, THREADS_X),
                DIV_UP(params.lattice_height, THREADS_Y));
    dim3 threads_per_block(THREADS_X, THREADS_Y);
    const int reduce_blocks = MIN(DIV_UP(params.total_words, THREADS),
                                  (props.maxThreadsPerMultiProcessor / THREADS) * props.multiProcessorCount);

    CHECK_CUDA(hipMalloc(&d_spins, params.total_words * sizeof(*d_spins)))
    CHECK_CUDA(hipMemset(d_spins, 0, params.total_words * sizeof(*d_spins)))

    CHECK_CUDA(hipMalloc(&d_sum, 2 * sizeof(*d_sum)))

    d_black_tiles = d_spins;
    d_white_tiles = d_spins + params.total_words / 2;

    float *d_probabilities;
    CHECK_CUDA(hipMallocPitch(&d_probabilities, &params.pitch,
                               5 * sizeof(*d_probabilities), 2))

    CHECK_CUDA(hipEventCreate(&start))
    CHECK_CUDA(hipEventCreate(&stop))

    // words_per_row / 2 because words two 64 bit words are compacted into
    // one 128 bit word
    initialise_arrays<unsigned long long>(
            blocks, threads_per_block,
            params.seed, params.words_per_row / 2,
            d_black_tiles, d_white_tiles, percentage_up
    );

    CHECK_CUDA(hipSetDevice(0))
    CHECK_CUDA(hipDeviceSynchronize())

    mag_file.open("magnetisation_0.dat");
    int iteration;
    float global_market;
    CHECK_CUDA(hipEventRecord(start, nullptr))
    for(iteration = 0; iteration < total_updates; iteration++) {
        global_market = update(
            iteration, blocks, threads_per_block, reduce_blocks,
            d_black_tiles, d_white_tiles, d_sum, d_probabilities,
            spins_up, spins_down, params
        );
        mag_file << global_market << std::endl;

        // create a new file every FILE_ENTRY_LIMIT iterations
        if (iteration % FILE_ENTRY_LIMIT == 0 && (iteration)) {
            mag_file.close();
            mag_file.open("magnetisation_" + std::to_string(iteration) + ".dat");
        }

        //if (iteration % 50 == 0)
        //   dumpLattice(iteration, params.lattice_height, params.words_per_row,
        //              params.total_words, d_spins);
    }
    mag_file.close();
    CHECK_CUDA(hipEventRecord(stop, nullptr))
    CHECK_CUDA(hipEventSynchronize(stop))

    CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop))
    double spin_updates_per_nanosecond = static_cast<double>(params.total_words * SPIN_X_WORD) * iteration / (elapsed_time * 1.0E+6);
    std::cout << "Beta: " << beta << std::endl;
    std::cout << "Computation time: " << elapsed_time * 1.0E-3 << "s" << std::endl;
    std::cout << "Updates per ns: " << spin_updates_per_nanosecond << std::endl;
    CHECK_CUDA(hipFree(d_spins))
    CHECK_CUDA(hipFree(d_probabilities))
    CHECK_CUDA(hipFree(d_sum))
    CHECK_CUDA(hipSetDevice(0))
    CHECK_CUDA(hipDeviceReset())
    return 0;
}