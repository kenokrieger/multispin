#include "hip/hip_runtime.h"
/* Program to simulate the Bornholdt Ising Model (https://arxiv.org/pdf/cond-mat/0105224.pdf)
 *
 * The program reads in a configuration file "multising.conf" or first command line parameter. The
 * configuration file contains the parameter choice for the simulation.
 *
 * Example configuration file:
 *
 * lattice_height = 8192           # Lattice size
 * lattice_width = 8192            # Lattice size
 * total_updates = 10000           # Number of iterations to perform
 * seed = 1591361                  # Seed for the simulation
 * alpha = 128.00                  # Parameter of the model (coupling strength to the magnetisation)
 * j = 1.0                         # Parameter of the model (coupling strength to the neighbours)
 * beta = 1.0                      # Parameter of the model (pseudo-temperature)
 * init_up = 0.5                   # Percentage of spins initially pointing up
 * rng_offset = 124837             # Used to resume the simulation at given time point in combination with import (optional)
 * import = iteration_124837.dat   # Used to resume simulation with given state in file (optional)
 * export = final_state.dat        # Save the final configuration to file with specified name (optional)
 *
 * By default, the relative magnetisation will be saved in a file magnetisation_*.dat where a new file every
 * FILE_ENTRY_LIMIT (1e6) iterations will be created.
 * To save the lattice configuration edit the if clause in the main update loop.
 *
 * At the end of the simulation an additional line will be added to the configuration file denoting the
 * reached number of iterations.
 *
 * final_iteration = 10000
 *
 * This can be used together with the exported final_state to resume the simulation.
 *
 */
#include <cstdio>
#include <string>
#include <iostream>
#include <fstream>
#include <map>
#include <csignal>

#include "cudamacro.h"
#include "traders.cuh"

#pragma clang diagnostic push
#pragma ide diagnostic ignored "OCDFAInspection"
using std::string;

#define DIV_UP(a,b)  (((a) + ((b) - 1)) / (b))
#define MIN(a,b)	(((a) < (b)) ? (a) : (b))

#define THREADS (128)
#define BIT_X_SPIN (4)

#define THREADS_X (16)
#define THREADS_Y (16)

#define FILE_ENTRY_LIMIT (1000000)


// trim from left
inline string& ltrim(string& s, const char* t = " \t\n\r\f\v") {
    s.erase(0, s.find_first_not_of(t));
    return s;
}

// trim from right
inline string& rtrim(string& s, const char* t = " \t\n\r\f\v") {
    s.erase(s.find_last_not_of(t) + 1);
    return s;
}

// trim from left & right
inline string& trim(string& s, const char* t = " \t\n\r\f\v") {
    return ltrim(rtrim(s, t), t);
}

volatile sig_atomic_t flag_terminate = 0;
void sigint(int sig) {  // can be called asynchronously
    flag_terminate = 1; // set flag
}


/**
 * @brief Read in the simulation details from a configuration file
 *
 * Numerous parameters need to be passed to the simulation via a configuration file in which fields and values
 * are separated by an equal sign.
 *
 * @param config_filename  The name of the configuration file
 * @param delimiter  The delimiter, or "assigment operator", for fields and values. Defaults to "="
 * @return  A map containing the fields and values read from the configuration file
 *
 */
std::map<string, string> readConfigFile(const char* config_filename, const string& delimiter = "=") {
    std::ifstream config_file;
    config_file.open(config_filename);
    std::map<string, string> config;

    if (!config_file.is_open()) {
        std::cout << "Could not open config file '" << config_filename << "'" << std::endl;
        exit(EXIT_FAILURE);
    }

    int row = 0;
    string line;
    string key;

    while (getline(config_file, line)) {
        if (line[0] == '#' || line.empty()) continue;
        int delimiter_position = line.find(delimiter);

        for (int idx = 0; idx < delimiter_position; idx++) {
            if (line[idx] != ' ') key += line[idx];
        }

        string value = line.substr(delimiter_position + 1, line.length() - 1);
        config[key] = value;
        row++;
        key = "";
    }
    config_file.close();
    return config;
}


void validateGrid(const long long lattice_width, const long long lattice_height,
                  const int spin_x_word) {
    if (!lattice_width || (lattice_width % 2) || ((lattice_width / 2) % (2 * spin_x_word * THREADS_X))) {
        fprintf(stderr, "\nPlease specify an lattice_width multiple of %d\n\n",
                2 * spin_x_word * 2 * THREADS_X);
        exit(EXIT_FAILURE);
    }
    if (!lattice_height || (lattice_height % (THREADS_Y))) {
        fprintf(stderr, "\nPlease specify a lattice_height multiple of %d\n\n", THREADS_Y);
        exit(EXIT_FAILURE);
    }
}


hipDeviceProp_t identifyGpu() {
    hipDeviceProp_t props{};
    CHECK_CUDA(hipGetDeviceProperties(&props, 0))
    /*
    printf("\nUsing GPU: %s, %d SMs, %d th/SM max, CC %d.%d, ECC %s\n",
    props.name, props.multiProcessorCount,
    props.maxThreadsPerMultiProcessor,
    props.major, props.minor,
    props.ECCEnabled ? "on" : "off");
    */
    return props;
}


int main(int argc, char **argv) {
    unsigned long long *d_spins = nullptr;
    const int SPIN_X_WORD = (8 * sizeof(*d_spins)) / BIT_X_SPIN;
    unsigned long long *d_black_tiles;
    unsigned long long *d_white_tiles;
    unsigned long long *d_sum;

    string import_file;
    string export_file;
    bool read_from_file = false;
    bool dump_to_file = false;

    hipEvent_t start, stop;
    float elapsed_time;

    std::ofstream mag_file;
    Parameters params;

    const char *config_filename = (argc == 1) ? "multising.conf" : argv[1];
    std::map<string, string> config = readConfigFile(config_filename);

    params.lattice_height = std::stoll(config["lattice_height"]);
    params.lattice_width = std::stoll(config["lattice_width"]);
    params.seed = std::stoull(config["seed"]);
    const unsigned int total_updates = std::stoul(config["total_updates"]);
    float alpha = std::stof(config["alpha"]);
    float j = std::stof(config["j"]);
    float beta = std::stof(config["beta"]);
    float percentage_up = std::stof(config["init_up"]);

    if (config.count("rng_offset")) {
        params.rng_offset = std::stoull(config["rng_offset"]);
    } else {
        params.rng_offset = 0;
    }
    if (config.count("import")) {
        import_file = config["import"];
        trim(import_file);
        read_from_file = true;
        std::cout << "Using existing lattice state from file: " << import_file << std::endl;
    }
    if (config.count("export")) {
        export_file = config["export"];
        trim(export_file);
        dump_to_file = true;
    }

    params.reduced_alpha = -2.0f * beta * alpha;
    params.reduced_j = -2.0f * beta * j;

    validateGrid(params.lattice_width, params.lattice_height, SPIN_X_WORD);
    hipDeviceProp_t props = identifyGpu();

    params.words_per_row = (params.lattice_width / 2) / SPIN_X_WORD;
    params.total_words = 2ull * static_cast<size_t>(params.lattice_height) * params.words_per_row;

    // words_per_row / 2 because each entry in the array has two components
    dim3 blocks(DIV_UP(params.words_per_row / 2, THREADS_X),
                DIV_UP(params.lattice_height, THREADS_Y));
    dim3 threads_per_block(THREADS_X, THREADS_Y);
    const int reduce_blocks = MIN(DIV_UP(params.total_words, THREADS),
                                  (props.maxThreadsPerMultiProcessor / THREADS) * props.multiProcessorCount);

    CHECK_CUDA(hipMalloc(&d_spins, params.total_words * sizeof(*d_spins)))
    CHECK_CUDA(hipMemset(d_spins, 0, params.total_words * sizeof(*d_spins)))

    CHECK_CUDA(hipMalloc(&d_sum, 2 * sizeof(*d_sum)))

    d_black_tiles = d_spins;
    d_white_tiles = d_spins + params.total_words / 2;

    float *d_probabilities;
    CHECK_CUDA(hipMallocPitch(&d_probabilities, &params.pitch,
                               5 * sizeof(*d_probabilities), 2))

    CHECK_CUDA(hipEventCreate(&start))
    CHECK_CUDA(hipEventCreate(&stop))

    if (read_from_file) {
        std::cout << "Reading in lattice configuration..." << std::endl;
        readFromFileBinary(d_spins, import_file.c_str(), params.lattice_height,
                     params.words_per_row, params.total_words);
    } else {
        // words_per_row / 2 because words two 64 bit words are compacted into
        // one 128 bit word
        std::cout << "Initialising random lattice state..." << std::endl;
        initialiseArrays<unsigned long long>(
                blocks, threads_per_block,
                params.seed, params.words_per_row / 2,
                d_black_tiles, d_white_tiles, percentage_up
        );
    }

    CHECK_CUDA(hipSetDevice(0))
    CHECK_CUDA(hipDeviceSynchronize())

    mag_file.open("magnetisation_" + std::to_string(params.rng_offset) + ".dat");
    int iteration;
    float relative_magnetisation;
    signal(SIGINT, sigint);
    CHECK_CUDA(hipEventRecord(start, nullptr))
    for(iteration = params.rng_offset; iteration < total_updates; iteration++) {
        relative_magnetisation = update(
                iteration, blocks, threads_per_block, reduce_blocks,
                d_black_tiles, d_white_tiles, d_sum, d_probabilities,
                params
        );
        mag_file << relative_magnetisation << std::endl;

        // create a new file every FILE_ENTRY_LIMIT iterations
        if (iteration % FILE_ENTRY_LIMIT == 0 && iteration) {
            mag_file.close();
            mag_file.open("magnetisation_" + std::to_string(iteration) + ".dat");
        }

        if (iteration % 30000000 == 0 && iteration) {
            FILE *f = fopen("backup.info", "w");
            fprintf(f, "Backup of iteration %d\n", iteration);
            fclose(f);
            dumpLatticeBinary("lattice_backup.bin", params.lattice_height, params.words_per_row,
                              params.total_words, d_spins);
        }
        if (flag_terminate) {
            std::cout << "Received keyboard interrupt, exiting..." << std::endl;
            break;
        }
    }
    mag_file.close();
    CHECK_CUDA(hipEventRecord(stop, nullptr))
    CHECK_CUDA(hipEventSynchronize(stop))

    CHECK_CUDA(hipEventElapsedTime(&elapsed_time, start, stop))
    double spin_updates_per_nanosecond = static_cast<double>(params.total_words * SPIN_X_WORD) * iteration / (elapsed_time * 1.0E+6);
    std::cout << "Computation time: " << elapsed_time * 1.0E-3 << "s" << std::endl;
    std::cout << "Updates per ns: " << spin_updates_per_nanosecond << std::endl;
    if (dump_to_file) {
        std::cout << "Saving lattice state for reuse..." << std::endl;
        dumpLatticeBinary(export_file.c_str(), params.lattice_height, params.words_per_row,
                    params.total_words, d_spins);
    }
    FILE *fp = fopen(config_filename, "a");
    fprintf(fp, "final_iteration = %d\n", iteration);
    fclose(fp);
    CHECK_CUDA(hipFree(d_spins))
    CHECK_CUDA(hipFree(d_probabilities))
    CHECK_CUDA(hipFree(d_sum))
    CHECK_CUDA(hipSetDevice(0))
    CHECK_CUDA(hipDeviceReset())
    return 0;
}
